#include "hip/hip_runtime.h"
/* Grupo 21 A:
Fernando Gorodscy - 7152354
Leonardo Rebelo - 5897894
*/

#include "opencv2/opencv.hpp"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>

using namespace std;
using namespace cv;

#define nThreadsPorBloco 512

__global__ void blur( int *in_image, int *out_image, int *cols, int *rows) {
    
	int v, i, j, k, w;
	int media;

	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	i = offset/(*cols);
	j = offset - i*(*cols);
            
    media = 0;
    v = 0;

    for(k = -2; k <= 2; k++){
        for(w = -2; w <= 2; w++){
            if((i + k >= 0) && (i + k < *rows) && (j + w >= 0) && (j + w < *cols)){
                media += in_image[(i+k)*(*cols) + (j+w)];
                v++;
            }
        }
    }

    out_image[offset] = media/v;

	//out_image[offset] = 0;
}

int main(int argc, const char* argv[]){
   
    time_t inicioTempo = time(NULL);
    time_t tempo;

    //Matrizes que guardam os canais de cor
	Mat in_image = imread(argv[1], 1);
	Mat out_image = imread(argv[1], 1);

	// Alocacao de memoria no device
    int *dev_out_image[3];
	hipMalloc((void**)&dev_out_image[0], in_image.cols*in_image.rows*sizeof(int));
    hipMalloc((void**)&dev_out_image[1], in_image.cols*in_image.rows*sizeof(int));
    hipMalloc((void**)&dev_out_image[2], in_image.cols*in_image.rows*sizeof(int));


	int *dev_in_image[3];
	hipMalloc( (void**)&dev_in_image[0], in_image.cols*in_image.rows*sizeof(int));
    hipMalloc( (void**)&dev_in_image[1], in_image.cols*in_image.rows*sizeof(int));
    hipMalloc( (void**)&dev_in_image[2], in_image.cols*in_image.rows*sizeof(int));

	int *dev_rows, *dev_cols;
	hipMalloc( (void**)&dev_cols, sizeof(int));
	hipMalloc( (void**)&dev_rows, sizeof(int));

	

    
    // Alocacao de memoria no host
	int *int_out_image[3];
    int_out_image[0] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);
    int_out_image[1] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);
    int_out_image[2] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);

	int *int_in_image[3];
	int_in_image[0] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);
    int_in_image[1] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);
    int_in_image[2] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);

    tempo = time(NULL) - inicioTempo;
    //Arquivo salvo na memoria principal.
    //Copiando para memoria da placa...

	// Convert Mat to int**
    for(int i = 0; i < in_image.rows; i++){
        for(int j = 0; j < in_image.cols; j++){
            int_in_image[0][i * in_image.cols + j] = in_image.at<Vec3b>(i, j)[0];
            int_in_image[1][i * in_image.cols + j] = in_image.at<Vec3b>(i, j)[1];
            int_in_image[2][i * in_image.cols + j] = in_image.at<Vec3b>(i, j)[2];
        }
    }

    // copia as matrizes da memoria do host para o device
    //hipMemcpy( dev_out_image, &out_image, out_image.elemSize(), hipMemcpyHostToDevice );
	hipMemcpy( dev_in_image[0], int_in_image[0], in_image.cols*in_image.rows*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_in_image[1], int_in_image[1], in_image.cols*in_image.rows*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_in_image[2], int_in_image[2], in_image.cols*in_image.rows*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_out_image[0], int_out_image[0], in_image.cols*in_image.rows*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_out_image[1], int_out_image[1], in_image.cols*in_image.rows*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_out_image[2], int_out_image[2], in_image.cols*in_image.rows*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_cols, &in_image.cols, sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_rows, &in_image.rows, sizeof(int), hipMemcpyHostToDevice );

    //Vetor copiado para memoria da placa.
    //Aplicando filtro de blur...

    //Realiza o filtro blur em cada matriz
    blur<<<in_image.cols*in_image.rows/nThreadsPorBloco,nThreadsPorBloco>>>( dev_in_image[0], dev_out_image[0], dev_cols, dev_rows);
	blur<<<in_image.cols*in_image.rows/nThreadsPorBloco,nThreadsPorBloco>>>( dev_in_image[1], dev_out_image[1], dev_cols, dev_rows);
	blur<<<in_image.cols*in_image.rows/nThreadsPorBloco,nThreadsPorBloco>>>( dev_in_image[2], dev_out_image[2], dev_cols, dev_rows);

    tempo = time(NULL) - inicioTempo;
    //Filtro Aplicado.
    //Copiando vetor para memoria principal...

    // Copia de volta as matrizes da memoria do Device para o Host
    hipMemcpy( int_out_image[0], dev_out_image[0], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( int_out_image[1], dev_out_image[1], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( int_out_image[2], dev_out_image[2], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( int_in_image[0], dev_in_image[0], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( int_in_image[1], dev_in_image[1], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( int_in_image[2], dev_in_image[2], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );

    // Convert int to Mat
    for(int i = 0; i < in_image.rows; i++){
        for(int j = 0; j < in_image.cols; j++){
            out_image.at<Vec3b>(i, j)[0] = int_out_image[0][i * in_image.cols + j];
            out_image.at<Vec3b>(i, j)[1] = int_out_image[1][i * in_image.cols + j];
            out_image.at<Vec3b>(i, j)[2] = int_out_image[2][i * in_image.cols + j];
        }
    }

	imwrite(argv[2], out_image);

	for(int i = 0; i < out_image.rows; i++){
        for(int j = 0; j < out_image.cols; j++){
            in_image.at<Vec3b>(i, j)[0] = int_in_image[0][i * in_image.cols + j];
            in_image.at<Vec3b>(i, j)[1] = int_in_image[1][i * in_image.cols + j];
            in_image.at<Vec3b>(i, j)[2] = int_in_image[2][i * in_image.cols + j];
        }
    }

	imwrite(argv[1], in_image);

    tempo = time(NULL) - inicioTempo;
    //Liberando memoria...

    in_image.release();
    out_image.release();
    hipFree( dev_in_image );
    hipFree( dev_out_image );

    tempo = time(NULL) - inicioTempo;
    //Memoria liberada.
    
    return 0;
}
