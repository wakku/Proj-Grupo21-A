#include "hip/hip_runtime.h"
/* Grupo 21 A:
Fernando Gorodscy - 7152354
Leonardo Rebelo - 5897894
*/

#include "opencv2/opencv.hpp"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>

using namespace std;
using namespace cv;

#define nThreadsPorBloco 512

__global__ void blur( int *in_image, int *out_image, int *cols, int *rows) {
    
	int v, i, j, k, w;
	int mediaR, mediaG, mediaB;
    int imageSize = (*cols) * (*rows);

	int offset = threadIdx.x + blockIdx.x * blockDim.x;
	i = offset/(*cols);
	j = offset - i*(*cols);
            
    mediaR = 0;
    mediaG = 0;
    mediaB = 0;
    v = 0;

    for(k = -2; k <= 2; k++){
        for(w = -2; w <= 2; w++){
            if((i + k >= 0) && (i + k < *rows) && (j + w >= 0) && (j + w < *cols)){
                mediaR += in_image[(i+k)*(*cols) + (j+w)];
                mediaG += in_image[(i+k)*(*cols) + (j+w) + imageSize];
                mediaB += in_image[(i+k)*(*cols) + (j+w) + imageSize];
                v++;
            }
        }
    }

    out_image[offset] = mediaR/v;
    out_image[offset + imageSize] = mediaG/v;
    out_image[offset + 2*imageSize] = mediaB/v;

	//out_image[offset] = 0;
}

int main(int argc, const char* argv[]){

    //Matrizes que guardam os canais de cor
	Mat in_image = imread(argv[1], 1);
	Mat out_image = imread(argv[1], 1);

	// Alocacao de memoria no device
    int *dev_out_image[1];
	hipMalloc((void**)&dev_out_image[0], in_image.cols*in_image.rows*sizeof(int)*3);


	int *dev_in_image[1];
	hipMalloc( (void**)&dev_in_image[0], in_image.cols*in_image.rows*sizeof(int)*3);

	int *dev_rows, *dev_cols;
	hipMalloc( (void**)&dev_cols, sizeof(int));
	hipMalloc( (void**)&dev_rows, sizeof(int));
    
    // Alocacao de memoria no host
	int *int_out_image[1];
    int_out_image[0] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows*3);

	int *int_in_image[1];
	int_in_image[0] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows*3);

    //Arquivo salvo na memoria principal.
    //Copiando para memoria da placa...

    int imageSize = in_image.rows * in_image.cols;
	// Convert Mat to int**
    for(int i = 0; i < in_image.rows; i++){
        for(int j = 0; j < in_image.cols; j++){
            int_in_image[0][i * in_image.cols + j] = in_image.at<Vec3b>(i, j)[0];
            int_in_image[0][i * in_image.cols + j + imageSize] = in_image.at<Vec3b>(i, j)[1];
            int_in_image[0][i * in_image.cols + j + 2*imageSize] = in_image.at<Vec3b>(i, j)[2];
        }
    }

    // copia as matrizes da memoria do host para o device
    //hipMemcpy( dev_out_image, &out_image, out_image.elemSize(), hipMemcpyHostToDevice );
	hipMemcpy( dev_in_image[0], int_in_image[0], in_image.cols*in_image.rows*sizeof(int)*3, hipMemcpyHostToDevice );
	hipMemcpy( dev_out_image[0], int_out_image[0], in_image.cols*in_image.rows*sizeof(int)*3, hipMemcpyHostToDevice );
	hipMemcpy( dev_cols, &in_image.cols, sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( dev_rows, &in_image.rows, sizeof(int), hipMemcpyHostToDevice );

    //Vetor copiado para memoria da placa.
    //Aplicando filtro de blur...

    //Realiza o filtro blur em cada matriz
    blur<<<in_image.cols*in_image.rows/nThreadsPorBloco,nThreadsPorBloco>>>( dev_in_image[0], dev_out_image[0], dev_cols, dev_rows);

    //Filtro Aplicado.
    //Copiando vetor para memoria principal...

    // Copia de volta as matrizes da memoria do Device para o Host
    hipMemcpy( int_out_image[0], dev_out_image[0], in_image.cols*in_image.rows*sizeof(int)*3, hipMemcpyDeviceToHost );
	hipMemcpy( int_in_image[0], dev_in_image[0], in_image.cols*in_image.rows*sizeof(int)*3, hipMemcpyDeviceToHost );

    // Convert int to Mat
    for(int i = 0; i < in_image.rows; i++){
        for(int j = 0; j < in_image.cols; j++){
            out_image.at<Vec3b>(i, j)[0] = int_out_image[0][i * in_image.cols + j];
            out_image.at<Vec3b>(i, j)[1] = int_out_image[0][i * in_image.cols + j + imageSize];
            out_image.at<Vec3b>(i, j)[2] = int_out_image[0][i * in_image.cols + j + 2*imageSize];
        }
    }

	imwrite(argv[2], out_image);

    //Liberando memoria...

    in_image.release();
    out_image.release();
    hipFree( dev_in_image );
    hipFree( dev_out_image );

    //Memoria liberada.
    
    return 0;
}
