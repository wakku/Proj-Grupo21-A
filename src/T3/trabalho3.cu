#include "hip/hip_runtime.h"
/* Grupo 21 A:
Fernando Gorodscy - 7152354
Leonardo Rebelo - 5897894
*/

#include "opencv2/opencv.hpp"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>

using namespace std;
using namespace cv;

#define nThreadsPorBloco 512

__host__ __device__ void teste(Mat *out_image, float media_R, float media_G, float media_B, int v, int i, int j)
{
	out_image->at<Vec3b>(i, j)[0] = media_R/v;
	out_image->at<Vec3b>(i, j)[1] = media_G/v;
    out_image->at<Vec3b>(i, j)[2] = media_B/v;
}


__global__ void blur( Mat *in_image, Mat *out_image) {
    
	uint8_t* pixelPtr = (uint8_t*)in_image->data;
	int v, i, j, k, w;        
	float media_R, media_G, media_B;

    for(i = 0; i < in_image->rows; i++){
        for(j = 0; j < in_image->cols; j++){
            
            media_R = 0;
            media_G = 0;
            media_B = 0;
            v = 0;

            for(k = -2; k <= 2; k++){
                for(w = -2; w <= 2; w++){
                    if((i + k >= 0) && (i + k < in_image->rows) && (j + w >= 0) && (j + w < in_image->cols)){
                        media_R += pixelPtr[(i+k)*in_image->cols + (j+w) + 0];
                        media_G += pixelPtr[(i+k)*in_image->cols + (j+w) + 1];
                        media_B += pixelPtr[(i+k)*in_image->cols + (j+w) + 2];
                        v++;
                    }
                }
            }
			

			teste(out_image, media_R, media_G, media_B, v, i, j); //isso foi um teste que nao deu certo, usando function global dentro de
 			//outra global
            //out_image->at<Vec3b>(i, j)[0] = media_R/v;
            //out_image->at<Vec3b>(i, j)[1] = media_G/v;
            //out_image->at<Vec3b>(i, j)[2] = media_B/v;
        }
    }
}

int main(int argc, const char* argv[]){
   
    time_t inicioTempo = time(NULL);
    time_t tempo;

    //Matrizes que guardam os canais de cor
	Mat in_image;
	Mat out_image;

    in_image = imread(argv[0], 1);
    out_image = imread(argv[0], 1);

    Mat *dev_out_image;
    Mat *dev_in_image;

    // Alocacao de memoria no device
    hipMalloc( (void**)&dev_out_image, in_image.elemSize());
    hipMalloc( (void**)&dev_in_image, in_image.elemSize());

	memset (&dev_out_image,0,fullSize);
    memset (&dev_in_image,0,fullSize);

    tempo = time(NULL) - inicioTempo;
    printf("Arquivo salvo na memoria principal.\n");
    printf("%ld : Copiando para memoria da placa...\n", tempo);

    // copia as matrizes da memoria do host para o device
    hipMemcpy( dev_in_image, &in_image, in_image.elemSize(), hipMemcpyHostToDevice );
    hipMemcpy( dev_out_image, &out_image, in_image.elemSize(), hipMemcpyHostToDevice );

    printf("%ld : Vetor copiado para memoria da placa.\n", tempo);
    printf("Aplicando filtro de blur...\n");

    //Realiza o filtro blur em cada matriz
    blur<<<in_image.elemSize()/nThreadsPorBloco,nThreadsPorBloco>>>( dev_in_image, dev_out_image);

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Filtro Aplicado.\n",tempo);
    printf("Copiando vetor para memoria principal...\n ");

    // Copia de volta as matrizes da memoria do Device para o Host
    hipMemcpy( &out_image, dev_out_image, in_image.elemSize(), hipMemcpyDeviceToHost );

    imwrite(argv[1], out_image);

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Arquivo salvo em out.ppm\n",tempo);
    printf("Liberando memoria..\n");

    in_image.release();
    out_image.release();
    hipFree( dev_in_image );
    hipFree( dev_out_image );

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Memoria liberada.\n",tempo);
    
    return 0;
}
