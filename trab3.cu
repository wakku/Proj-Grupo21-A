#include "hip/hip_runtime.h"
/* Grupo 21 A:
Fernando Gorodscy - 7152354
Leonardo Rebelo - 5897894
*/

#include "opencv2/opencv.hpp"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>

using namespace std;
using namespace cv;

#define nThreadsPorBloco 512

__global__ void blur( Mat *in_image, int *out_image[3]) {
    
	uint8_t* pixelPtr = (uint8_t*)in_image->data;
	int v, i, j, k, w;        
	float media_R, media_G, media_B;

    for(i = 0; i < in_image->rows; i++){
        for(j = 0; j < in_image->cols; j++){
            
            media_R = 0;
            media_G = 0;
            media_B = 0;
            v = 0;

            for(k = -2; k <= 2; k++){
                for(w = -2; w <= 2; w++){
                    if((i + k >= 0) && (i + k < in_image->rows) && (j + w >= 0) && (j + w < in_image->cols)){
                        media_R += pixelPtr[(i+k)*in_image->cols + (j+w) + 0];
                        media_G += pixelPtr[(i+k)*in_image->cols + (j+w) + 1];
                        media_B += pixelPtr[(i+k)*in_image->cols + (j+w) + 2];
                        v++;
                    }
                }
            }
	
            out_image[0][i * in_image->cols + j] = media_R/v;
            out_image[1][i * in_image->cols + j] = media_G/v;
            out_image[2][i * in_image->cols + j] = media_B/v;
        }
    }
}

int main(int argc, const char* argv[]){
   
    time_t inicioTempo = time(NULL);
    time_t tempo;

    //Matrizes que guardam os canais de cor
	Mat in_image;
	Mat out_image;
    int *int_out_image[3];

    in_image = imread(argv[1], 1);
    out_image = imread(argv[1], 1);

    int *dev_out_image[3];
    Mat *dev_in_image;

    // Alocacao de memoria no device
    hipMalloc( (void**)&dev_in_image, in_image.elemSize());
    hipMalloc( (void**)&dev_out_image[0], in_image.cols*in_image.rows*sizeof(int));
    hipMalloc( (void**)&dev_out_image[1], in_image.cols*in_image.rows*sizeof(int));
    hipMalloc( (void**)&dev_out_image[2], in_image.cols*in_image.rows*sizeof(int));

    int_out_image[0] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);
    int_out_image[1] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);
    int_out_image[2] = (int*) malloc(sizeof(int)*in_image.cols*in_image.rows);

    memset (&dev_in_image,0,in_image.elemSize());
    memset (int_out_image[0],0,sizeof(int)*in_image.cols*in_image.rows);
    memset (int_out_image[1],0,sizeof(int)*in_image.cols*in_image.rows);
    memset (int_out_image[2],0,sizeof(int)*in_image.cols*in_image.rows);

    tempo = time(NULL) - inicioTempo;
    printf("Arquivo salvo na memoria principal.\n");
    printf("%ld : Copiando para memoria da placa...\n", tempo);

    // copia as matrizes da memoria do host para o device
    hipMemcpy( dev_in_image, &in_image, in_image.elemSize(), hipMemcpyHostToDevice );

    printf("%ld : Vetor copiado para memoria da placa.\n", tempo);
    printf("Aplicando filtro de blur...\n");

    //Realiza o filtro blur em cada matriz
    blur<<<in_image.elemSize()/nThreadsPorBloco,nThreadsPorBloco>>>( dev_in_image, dev_out_image);

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Filtro Aplicado.\n",tempo);
    printf("Copiando vetor para memoria principal...\n ");

    // Copia de volta as matrizes da memoria do Device para o Host
    hipMemcpy( int_out_image[0], dev_out_image[0], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( int_out_image[1], dev_out_image[1], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );
    hipMemcpy( int_out_image[2], dev_out_image[2], in_image.cols*in_image.rows*sizeof(int), hipMemcpyDeviceToHost );

    // Convert int to Mat
    for(int i = 0; i < in_image.rows; i++){
        for(int j = 0; j < in_image.cols; j++){
    
            out_image.at<Vec3b>(i, j)[0] = int_out_image[0][i * in_image.cols + j];
            out_image.at<Vec3b>(i, j)[1] = int_out_image[1][i * in_image.cols + j];
            out_image.at<Vec3b>(i, j)[2] = int_out_image[2][i * in_image.cols + j];
        }
    }

	imwrite(argv[2], out_image);

    tempo = time(NULL) - inicioTempo;
    cout << tempo << ": Arquivo salvo em " << argv[2] << endl;
    cout << "Liberando memoria..." << endl;

    in_image.release();
    out_image.release();
    hipFree( dev_in_image );
    hipFree( dev_out_image );

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Memoria liberada.\n",tempo);
    
    return 0;
}
