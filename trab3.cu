/* Grupo 21 A:
Fernando Gorodscy - 7152354
Leonardo Rebelo - 5897894
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>


#define sizeX 720  // Invertido
#define sizeY 1280
#define nThreadsPorBloco 512
#define imEntrada "HD.ppm"
#define imSaida "out_4k_2.ppm"


__global__ void blur( int *imagem_in, int *imagem_out, int colorOffset ) {

    int offset = threadIdx.x + blockIdx.x * blockDim.x;

    int media=0;

    int cx,cy;

 	//Calcula a media para o pixel do centro
    for (cx = -2; cx <= 2; cx++){
        for (cy = -2; cy <= 2; cy++){
            media = media + imagem_in[colorOffset + offset + cx+ sizeX*cy];
        }
    }

    media = media/25;

    cx = 0, cy = 0;

    imagem_out[ colorOffset + offset ] =  media;
}

int main()
{   
    time_t inicioTempo = time(NULL);
    time_t tempo;

    int offsetR = 0;
    int offsetG = sizeX*sizeY;
    int offsetB = sizeX*sizeY*2;

    int i;
    int size = sizeof(int)*sizeX*sizeY;
    int fullSize = 3*size;

    //Vetor que guarda os canais de cor
    int *imagemRGB = (int *) malloc(fullSize); 
    int *outimagemRGB = (int *) malloc(fullSize);


    int *dev_outimagemRGB;
    int *dev_imagemRGB;

    // Alocacao de memoria no device
    hipMalloc( (void**)&dev_outimagemRGB, fullSize);
    hipMalloc( (void**)&dev_imagemRGB, fullSize);

    // Limpa os enderecos alocados para as matrizes
    memset (outimagemRGB,0,fullSize);
    memset (imagemRGB,0,fullSize);

    char linha[1000];
    int valor; //Pega o valor da linha
    int countCor = 0; // 0->R  1->G  2->B
    int ri = 0; //Contadores individuais das matrizes de cor
    int rj = 0;
    int gi = 0;
    int gj = 0;
    int bi = 0;
    int bj = 0;

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Abrindo arquivo e carregando na memoria...\n", tempo);

    // Abre o arquivo
    FILE * file;
    file = fopen( imEntrada  , "r");

    //Pula as 4 primeiras linhas, que sao comentarios e informacoes da imagem
    for ( i = 0; i < 4; i ++)
        fgets(linha, 1000, file);

    //Quebra a imagem ppm em no vetor
    while ( fgets(linha, 10, file)  != NULL ) {

        //Converte string para inteiro
        valor = atoi(linha); 

        //Verifica se chegou no no ultimo pixel de X e incrementa Y
        if(bi == sizeY){
            ri = 0;
            rj++;
            gi = 0;
            gj++;
            bi = 0;
            bj++;
            //printf("\n");
        }

         // How to do it.
         // [j,i]
         // j + sizeX*i

        if(countCor == 0){ //Salva em RED
            imagemRGB[offsetR + rj+ sizeX*ri] = valor;
            countCor = 1;
            ri++;
            continue;
        }
        if(countCor == 1){ //Salva em GREEN
            imagemRGB[offsetG +  gj+ sizeX*gi] = valor;
            countCor = 2;
            gi++;
            continue;
        }
        if(countCor == 2){ //Salva em BLUE
            imagemRGB[offsetB + bj+ sizeX*bi] = valor;
            countCor = 0;
            bi++;
            continue;
        } 
    }

    fclose(file);

    tempo = time(NULL) - inicioTempo;
    printf("Arquivo salvo na memoria principal.\n");
    printf("%ld : Copiando para memoria da placa...\n", tempo);

    // copia as matrizes da memoria do host para o device
    hipMemcpy( dev_imagemRGB, imagemRGB, fullSize, hipMemcpyHostToDevice );
    hipMemcpy( dev_outimagemRGB, outimagemRGB, fullSize, hipMemcpyHostToDevice );


    printf("%ld : Vetor copiado para memoria da placa.\n", tempo);
    printf("Aplicando filtro de blur...\n");

    //Realiza o filtro blur em cada matriz
    blur<<<sizeX*sizeY/nThreadsPorBloco,nThreadsPorBloco>>>( dev_imagemRGB, dev_outimagemRGB, offsetR);
    blur<<<sizeX*sizeY/nThreadsPorBloco,nThreadsPorBloco>>>( dev_imagemRGB, dev_outimagemRGB, offsetG);
    blur<<<sizeX*sizeY/nThreadsPorBloco,nThreadsPorBloco>>>( dev_imagemRGB, dev_outimagemRGB, offsetB);

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Filtro Aplicado.\n",tempo);
    printf("Copiando vetor para memoria principal...\n ");

    // Copia de volta as matrizes da memoria do Device para o Host
    hipMemcpy( outimagemRGB, dev_outimagemRGB, fullSize, hipMemcpyDeviceToHost );

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Vetor copiado para memoria principal...\n", tempo);
    printf("Salvando vetor no arquivo...\n");

    //Salva de volta como arquivo ppm
    // Abrir o arquivo
    FILE * out;
    out = fopen( imSaida , "w+");
    char saida[1000];
    char aux[100];

    //Coloca as 4 primeiras linhas:
    fputs("P3\n",out);

    sprintf(saida,"%d %d\n",sizeY,sizeX);
    fputs(saida,out);

    sprintf(saida,"# Imagem dos Brother\n");
    fputs(saida,out);

    fputs("255\n",out);

    //Coloca as matrizes de cor na saida ppm
    countCor = 0;
    ri = 0;
    rj = 0;
    gi = 0;
    gj = 0;
    bi = 0;
    bj = 0;



    while ( rj < sizeX && ri < sizeX*3 ){

        if(bi == sizeY){
            ri = 0;
            rj++;
            gi = 0;
            gj++;
            bi = 0;
            bj++;
        }

        if(countCor == 0){ //Salva em RED
            sprintf(aux,"%d",outimagemRGB[offsetR +rj+ sizeX*ri]);
            strcpy (saida,aux);
            strcat(saida,"\n");
            fputs(saida,out);
            countCor = 1;
            ri++;
            continue;
        }
        if(countCor == 1){ //Salva em GREEN
            sprintf(aux,"%d",outimagemRGB[offsetG+ gj+ sizeX*gi]);
            strcpy (saida,aux);
            strcat(saida,"\n");
            fputs(saida,out);
            countCor = 2;
            gi++;
            continue;
        }
        if(countCor == 2){ //Salva em BLUE
            sprintf(aux,"%d",outimagemRGB[offsetB +bj+ sizeX*bi]);
            strcpy (saida,aux);
            strcat(saida,"\n");
            fputs(saida,out);
            countCor = 0;
            bi++;
            continue;
        }



    }

    fclose(out);

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Arquivo salvo em out.ppm\n",tempo);
    printf("Liberando memoria..\n");

    free( imagemRGB );
    free( outimagemRGB );
    hipFree( dev_imagemRGB );
    hipFree( dev_outimagemRGB );

    tempo = time(NULL) - inicioTempo;
    printf("%ld : Memoria liberada.\n",tempo);
    

    return 0;
}
